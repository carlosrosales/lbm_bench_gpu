#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------------
// Function : init_f
// Revision : 1.0 (2016/02/22)
// Author   : Carlos Rosales-Fernandez [carlos.rosales.fernandez(at)gmail.com]
//-------------------------------------------------------------------------------
// Initialize all variables and arrays.
// This function requires 15 arguments.
//-------------------------------------------------------------------------------

//-------------------------------------------------------------------------------
// Copyright 2016 Carlos Rosales Fernandez and The University of Texas at Austin.
// Copyright 2008 Carlos Rosales Fernandez, David S. Whyte and IHPC (A*STAR).
//
// This file is part of MP-LABS.
//
// MP-LABS is free software: you can redistribute it and/or modify it under the
// terms of the GNU GPL version 3 or (at your option) any later version.
//
// MP-LABS is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
// A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License along with
// MP-LABS, in the file COPYING.txt. If not, see <http://www.gnu.org/licenses/>.
//-------------------------------------------------------------------------------

__global__ void init_f( int *nb_east_d,  int *nb_west_d,
                        int *nb_north_d, int *nb_south_d,
                        float *phi_d,    float *f_0_d,
                        float *f_1_d,    float *f_2_d,
                        float *f_3_d,    float *f_4_d,
                        float *f_5_d,    float *f_6_d )
{
    int   i, idx, ie, iw, j, jn, js, k, kt, kb;
    float Af, muPhin, phin, lapPhi;

    // Identify current thread
	  i = blockIdx.x * blockDim.x + threadIdx.x;
	  j = blockIdx.y * blockDim.y + threadIdx.y;

    // Initialize distribution function f
    for( k = 1; k < NZ-1; k++ ){

        // Define some local values
        idx  = gridId( i, j, k );
        phin = phi_d[idx];

        // Differential terms
        ie = nb_east_d[idx];
        iw = nb_west_d[idx];
        jn = nb_north_d[idx];
        js = nb_south_d[idx];
        kt = k + 1;
        kb = k - 1;

        // Laplacian of the order parameter Phi
        lapPhi = ( phi_d[ gridId(ie,jn,k ) ] + phi_d[ gridId(iw,js,k ) ] 
               +   phi_d[ gridId(ie,js,k ) ] + phi_d[ gridId(iw,jn,k ) ]
               +   phi_d[ gridId(ie,j ,kt) ] + phi_d[ gridId(iw,j ,kb) ]
               +   phi_d[ gridId(ie,j ,kb) ] + phi_d[ gridId(iw,j ,kt) ]
               +   phi_d[ gridId(i ,jn,kt) ] + phi_d[ gridId(i ,js,kb) ]
               +   phi_d[ gridId(i ,jn,kb) ] + phi_d[ gridId(i ,js,kt) ]
               + 2.f*( phi_d[ gridId(ie,j ,k ) ] + phi_d[ gridId(iw,j ,k ) ] 
               +       phi_d[ gridId(i ,jn,k ) ] + phi_d[ gridId(i ,js,k ) ] 
               +       phi_d[ gridId(i ,j ,kt) ] + phi_d[ gridId(i ,j ,kb) ]
               -       12.f*phin ) )*inv6;

         // Chemical potential
         muPhin = alpha4_d*phin*( phin*phin - phiStarSq_d ) - kappa_d*lapPhi;

        // Set distribution function f to its equilibrium value
        Af = 0.5f*Gamma_d*muPhin;

        f_0_d[idx] = phin - 3.f*Gamma_d*muPhin;

        f_1_d[idx] = Af;  f_2_d[idx] = Af;
        f_3_d[idx] = Af;  f_4_d[idx] = Af;
        f_5_d[idx] = Af;  f_6_d[idx] = Af;
    }
}

