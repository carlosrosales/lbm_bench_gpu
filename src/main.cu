//-------------------------------------------------------------------------------
// Program   : lbm_bench_gpu
// Revision  : 1.0 (2016/02/22)
// Author    : Carlos Rosales-Fernandez [carlos.rosales.fernandez(at)gmail.com]
//-------------------------------------------------------------------------------
// Driver for the hybrid MPI+OpenMP implementation of the Zheng-Shu-Chew
// multiphase LBM using D3Q7/D3Q19 discretization and periodic boundary
// conditions, including the gravitational force. For details:
//
// Journal of Computational Physics 218: 353-371, 2006.
//
// The average velocity, mass conservation factor, effective radius of the drop,
// pressure difference between the inside and the outside of the drop and the
// error with respect to the analytical value given by Laplace's equation are
// written to file "stats.out"
//
// The values of RELAX are used to control output:
// RELAX = -1 -> Setup Stage Completed
// RELAX =  0 -> Relaxation Stage Completed
// RELAX =  1 -> Main Calculation Stage Completed
//-------------------------------------------------------------------------------

//-------------------------------------------------------------------------------
// Copyright 2016 Carlos Rosales Fernandez and The University of Texas at Austin.
// Copyright 2008 Carlos Rosales Fernandez, David S. Whyte and IHPC (A*STAR).
//
// This file is part of MP-LABS.
//
// MP-LABS is free software: you can redistribute it and/or modify it under the
// terms of the GNU GPL version 3 or (at your option) any later version.
//
// MP-LABS is distributed in the hope that it will be useful, but WITHOUT ANY
// WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
// A PARTICULAR PURPOSE. See the GNU General Public License for more details.
//
// You should have received a copy of the GNU General Public License along with
// MP-LABS, in the file COPYING.txt. If not, see <http://www.gnu.org/licenses/>.
//-------------------------------------------------------------------------------

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <mpi.h>
#include <hip/hip_runtime.h>
#include "constants.h"

// CPU functions
#include "gridid.c"
#include "init.c"
#include "paramread.c"
#include "logsave.c"
#include "stats.c"
#include "mpiupdate.c"
#include "mpiupdate_f.c"
#include "mpiupdate_phi.c"
#include "vgrid.c"
#include "vtksave.c"

// GPU kernels
#include "init_f.cu"
#include "init_g.cu"
#include "update_phi.cu"
#include "update_rho.cu"
#include "update_velocity.cu"
#include "collision_f.cu"
#include "collision_g.cu"
#include "pack_mpi.cu"
#include "pack_mpi_f.cu"
#include "pack_mpi_phi.cu"
#include "stream_f.cu"
#include "stream_g.cu"
#include "unpack_mpi.cu"
#include "unpack_mpi_f.cu"
#include "unpack_mpi_phi.cu"

void fatalError( char *errorStr )
{
    if( proc == 0 ) fprintf( stderr, "ERROR: %s.\n", errorStr );
    MPI_Abort( MPI_COMM_WORLD, -1 );
}


int main( int argc, char **argv )
{
    int   device;
    float cpuMem, devMem, distMem, gpuMem, hydroMem, mpiMem, nbMem;
    char devName[32];

    // Initialize MPI environment
    MPI_Init( &argc, &argv );
    MPI_Comm_size( MPI_COMM_WORLD, &nprocs );
    MPI_Comm_rank( MPI_COMM_WORLD, &proc );

    // choose the GPU for execution
    device = proc % 2;
    hipSetDevice(device);
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);

    // Declare and create timing events
    hipEvent_t start, stopSetup, stopRelax, stopMain;
    hipEventCreate( &start );
    hipEventCreate( &stopSetup  );
    hipEventCreate( &stopRelax  );
    hipEventCreate( &stopMain  );

    // Write some information about the card we are runing on
    // Assume all cards in the cluster are the same
    if( proc == 0 ){
        fprintf( stdout, "*** Multiphase Zheng-Shu-Chew LBM 3D Simulation \n" );
        fprintf( stdout, "*** CUDA MPI Implementation Version 12.0-devel  \n" );
        fprintf( stdout, "\n" );

        strncpy( devName, properties.name, 32 );
        devMem = properties.totalGlobalMem;
        fprintf( stdout, "*** Program running on : %s\n", devName );
        if( devMem >= GB ){
            fprintf( stdout, "*** Total GPU Memory   : %.4f GB\n\n", devMem / GB);
        }else{
            fprintf( stdout, "*** Total GPU Memory   : %.4f MB\n\n", devMem / MB);
        }
    }

//====================== ARRAY DECLARATIONS ====================================
    // Host array declarations. These arrays have dinensions NX*NY*NZ, and
    // can be traversed using the running index idx = j + NY*i
    int   *nb_east = 0, *nb_west = 0, *nb_north  = 0, *nb_south  = 0;

    float *top_snd  = 0, *bot_snd  = 0, *top_rcv  = 0, *bot_rcv  = 0;
    float *topF_snd = 0, *botF_snd = 0, *topF_rcv = 0, *botF_rcv = 0;

    float *phi = 0, *rho = 0, *ux = 0, *uy = 0, *uz = 0;

    // Device array declarations. These arrays have dinensions NX*NY*NZ, and
    // can be traversed using the running index idx = k + NZ*( j + NY*i )
    int   *nb_east_d = 0, *nb_west_d = 0, *nb_north_d  = 0, *nb_south_d  = 0;

    float *top_snd_d  = 0, *bot_snd_d  = 0, *top_rcv_d  = 0, *bot_rcv_d  = 0;
    float *topF_snd_d = 0, *botF_snd_d = 0, *topF_rcv_d = 0, *botF_rcv_d = 0;

    float *phi_d = 0, *rho_d = 0, *ux_d = 0, *uy_d = 0, *uz_d = 0;

    float *f_0_d = 0, *f_1_d = 0, *f_2_d = 0, *f_3_d = 0, *f_4_d = 0,
          *f_5_d = 0, *f_6_d = 0;

    float *g_0_d  = 0, *g_1_d  = 0, *g_2_d  = 0, *g_3_d  = 0, *g_4_d  = 0,
          *g_5_d  = 0, *g_6_d  = 0, *g_7_d  = 0, *g_8_d  = 0, *g_9_d  = 0,
          *g_10_d = 0, *g_11_d = 0, *g_12_d = 0, *g_13_d = 0, *g_14_d = 0,
          *g_15_d = 0, *g_16_d = 0, *g_17_d = 0, *g_18_d = 0;


//====================== READ INPUT PARAMETERS =================================
    hipEventRecord( start, 0 );
    hipEventSynchronize( start );
    RELAX = -1;
    paramRead();
    vgrid();

    NX_h  = xmax;
    NY_h  = ymax;
    NZ_h  = ( zug - zlg + 1 );
    NXY_h = NX_h * NY_h;
    bufSize  = NX_h*NY_h;
    gridSize = NX_h*NY_h*NZ_h;

    // Sanity check for domain size
    if( (NX_h%32) != 0 ) fatalError( "NX is not a multiple of 32");
    if( (NY_h%32) != 0 ) fatalError( "NY is not a multiple of 32");

    // Sanity check for partition configuration
    if( BLOCK_SIZE_X*BLOCK_SIZE_Y > 512 ) fatalError( "Too many blocks" );
    if( NX_h < BLOCK_SIZE_X ) fatalError( "BLOCK_SIZE_X is too large");
    if( NY_h < BLOCK_SIZE_Y ) fatalError( "BLOCK_SIZE_Y is too large");

    // Define the grid and the number of threads per block for the calculations
    dim3 dimblock( BLOCK_SIZE_X, BLOCK_SIZE_Y, 1 );
    dim3 dimgrid( NX_h/BLOCK_SIZE_X, NY_h/BLOCK_SIZE_Y, 1 );

//====================== HOST ARRAY MEMORY ALLOCATION ==========================
    nb_east  = (int *)malloc( gridSize*sizeof(int) );
    nb_west  = (int *)malloc( gridSize*sizeof(int) );
    nb_north = (int *)malloc( gridSize*sizeof(int) );
    nb_south = (int *)malloc( gridSize*sizeof(int) );

    top_snd = (float *)malloc( 6*bufSize*sizeof(float) );
    top_rcv = (float *)malloc( 6*bufSize*sizeof(float) );
    bot_snd = (float *)malloc( 6*bufSize*sizeof(float) );
    bot_rcv = (float *)malloc( 6*bufSize*sizeof(float) );

    topF_snd = (float *)malloc( bufSize*sizeof(float) );
    topF_rcv = (float *)malloc( bufSize*sizeof(float) );
    botF_snd = (float *)malloc( bufSize*sizeof(float) );
    botF_rcv = (float *)malloc( bufSize*sizeof(float) );

    phi = (float *)malloc( gridSize*sizeof(float) );
    rho = (float *)malloc( gridSize*sizeof(float) );

    ux = (float *)malloc( gridSize*sizeof(float) );
    uy = (float *)malloc( gridSize*sizeof(float) );
    uz = (float *)malloc( gridSize*sizeof(float) );

//====================== DEVICE ARRAY MEMORY ALLOCATION ========================
    hipMalloc( (void **) &nb_east_d,  gridSize*sizeof(int) );
    hipMalloc( (void **) &nb_west_d,  gridSize*sizeof(int) );
    hipMalloc( (void **) &nb_north_d, gridSize*sizeof(int) );
    hipMalloc( (void **) &nb_south_d, gridSize*sizeof(int) );

    hipMalloc( (void **) &top_snd_d, 6*bufSize*sizeof(float) );
    hipMalloc( (void **) &top_rcv_d, 6*bufSize*sizeof(float) );
    hipMalloc( (void **) &bot_snd_d, 6*bufSize*sizeof(float) );
    hipMalloc( (void **) &bot_rcv_d, 6*bufSize*sizeof(float) );

    hipMalloc( (void **) &topF_snd_d, bufSize*sizeof(float) );
    hipMalloc( (void **) &topF_rcv_d, bufSize*sizeof(float) );
    hipMalloc( (void **) &botF_snd_d, bufSize*sizeof(float) );
    hipMalloc( (void **) &botF_rcv_d, bufSize*sizeof(float) );

    hipMalloc( (void **) &phi_d, gridSize*sizeof(float) );
    hipMalloc( (void **) &rho_d, gridSize*sizeof(float) );

    hipMalloc( (void **) &ux_d, gridSize*sizeof(float) );
    hipMalloc( (void **) &uy_d, gridSize*sizeof(float) );
    hipMalloc( (void **) &uz_d, gridSize*sizeof(float) );

    hipMalloc( (void **) &f_0_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &f_1_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &f_2_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &f_3_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &f_4_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &f_5_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &f_6_d, 2*gridSize*sizeof(float) );

    hipMalloc( (void **) &g_0_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_1_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_2_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_3_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_4_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_5_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_6_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_7_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_8_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_9_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_10_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_11_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_12_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_13_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_14_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_15_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_16_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_17_d, 2*gridSize*sizeof(float) );
    hipMalloc( (void **) &g_18_d, 2*gridSize*sizeof(float) );

//====================== DEVICE ARRAY ALLOCATION TEST ==========================
    if( nb_north_d == 0 || nb_south_d == 0 || nb_east_d  == 0 || nb_west_d == 0 )
        fatalError( "Unable to allocate memory for neighbors on device" );

    if( top_snd_d == 0  || bot_snd_d == 0  || top_rcv_d == 0  || 
        bot_rcv_d == 0  || topF_snd_d == 0 || topF_rcv_d == 0 ||
        botF_snd_d == 0 || botF_rcv_d == 0 )
        fatalError( "Unable to allocate memory for MPI buffers on device" );

    if( phi_d == 0 || rho_d == 0 || ux_d == 0 || uy_d == 0 || uz_d == 0 )
        fatalError( "Unable to allocate memory for hydro variables on device" );

    if( f_0_d == 0 || f_1_d == 0 || f_2_d == 0 || f_3_d == 0 || 
        f_4_d == 0 || f_5_d == 0 || f_6_d == 0)
        fatalError( "Unable to allocate memory for f on device" );

    if( g_0_d  == 0 || g_1_d  == 0 || g_2_d  == 0 || g_3_d  == 0 || 
        g_4_d  == 0 || g_5_d  == 0 || g_6_d  == 0 || g_7_d  == 0 ||
        g_8_d  == 0 || g_9_d  == 0 || g_10_d == 0 || g_11_d == 0 ||
        g_12_d == 0 || g_13_d == 0 || g_14_d == 0 || g_15_d == 0 ||
        g_16_d == 0 || g_17_d == 0 || g_18_d == 0 )
        fatalError( "Unable to allocate memory for g on device" );


    nbMem    = 4.f*sizeof(int)*gridSize;
    mpiMem   = 4.f*sizeof(float)*bufSize*7.f;
    hydroMem = 5.f*sizeof(float)*gridSize;
    distMem  = 2.f*( 19.f*sizeof(float)*gridSize + 7.f*sizeof(float)*gridSize );
    gpuMem   = nbMem + mpiMem + hydroMem + distMem;
    cpuMem   = nbMem + mpiMem + hydroMem;
    if( proc == master ){
        if( cpuMem >= GB )
            fprintf( stdout, "Allocated %.4f GB or memory on host.\n", cpuMem/GB );
        if( cpuMem < GB )
            fprintf( stdout, "Allocated %.4f MB or memory on host.\n", cpuMem/MB );
        if( gpuMem >= GB )
            fprintf( stdout, "Allocated %.4f GB or memory on device.\n", gpuMem/GB );
        if( gpuMem < GB )
            fprintf( stdout, "Allocated %.4f MB or memory on device.\n", gpuMem/MB );
    }

//====================== INITIALIZE DATA =======================================
    init( nb_east, nb_west, nb_north, nb_south, phi, rho, ux, uy, uz );
    logSave( devName, devMem, nb_east, nb_west, nb_north, nb_south, 
             phi, rho, ux, uy, uz );
    RELAX = 1;

    stats( phi, ux, uy, uz );

    vtkSave( nb_east, nb_west, nb_north, nb_south, phi, rho, ux, uy, uz );

    hipMemcpy( nb_east_d,  nb_east,  sizeof(int)*gridSize, hipMemcpyHostToDevice ); 
    hipMemcpy( nb_west_d,  nb_west,  sizeof(int)*gridSize, hipMemcpyHostToDevice );
    hipMemcpy( nb_north_d, nb_north, sizeof(int)*gridSize, hipMemcpyHostToDevice ); 
    hipMemcpy( nb_south_d, nb_south, sizeof(int)*gridSize, hipMemcpyHostToDevice );  

    hipMemcpy( phi_d, phi, sizeof(float)*gridSize, hipMemcpyHostToDevice );
    hipMemcpy( rho_d, rho, sizeof(float)*gridSize, hipMemcpyHostToDevice );
    hipMemcpy( ux_d,  ux,  sizeof(float)*gridSize, hipMemcpyHostToDevice );
    hipMemcpy( uy_d,  uy,  sizeof(float)*gridSize, hipMemcpyHostToDevice );
    hipMemcpy( uz_d,  uz,  sizeof(float)*gridSize, hipMemcpyHostToDevice );

    hipMemcpyToSymbol(HIP_SYMBOL( dcol), &gridSize, sizeof(int), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( NX),   &NX_h,  sizeof(int), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( NY),   &NY_h,  sizeof(int), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( NZ),   &NZ_h,  sizeof(int), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( NXY),  &NXY_h, sizeof(int), 0, hipMemcpyHostToDevice );

    hipMemcpyToSymbol(HIP_SYMBOL( K0), &Wn0, sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( K1), &Wn1, sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( K2), &Wn2, sizeof(float), 0, hipMemcpyHostToDevice );

    hipMemcpyToSymbol(HIP_SYMBOL( KC0), &WnC0, sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( KC1), &WnC1, sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( KC2), &WnC2, sizeof(float), 0, hipMemcpyHostToDevice );

    hipMemcpyToSymbol(HIP_SYMBOL( alpha4_d),    &alpha4,    sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( kappa_d),     &kappa,     sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( Gamma_d),     &Gamma,     sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( eta_d),       &eta,       sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( eta2_d),      &eta2,      sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( phiStarSq_d), &phiStarSq, sizeof(float), 0, hipMemcpyHostToDevice );

    hipMemcpyToSymbol(HIP_SYMBOL( invEta2_d),      &invEta2,      sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( invTauPhi_d),    &invTauPhi,    sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( invTauPhiOne_d), &invTauPhiOne, sizeof(float), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( invTauRhoOne_d), &invTauRhoOne, sizeof(float), 0, hipMemcpyHostToDevice );

    hipMemcpyToSymbol(HIP_SYMBOL( zl_d), &zl, sizeof(int), 0, hipMemcpyHostToDevice );
    hipMemcpyToSymbol(HIP_SYMBOL( zu_d), &zu, sizeof(int), 0, hipMemcpyHostToDevice );

    init_f <<<dimgrid,dimblock>>> ( nb_east_d, nb_west_d, nb_north_d, 
                                    nb_south_d, phi_d, f_0_d, f_1_d, 
                                    f_2_d, f_3_d, f_4_d, f_5_d, f_6_d );

    init_g <<<dimgrid,dimblock>>> ( nb_east_d, nb_west_d, nb_north_d, 
                                    nb_south_d, phi_d, rho_d, g_0_d,
                                    g_1_d, g_2_d, g_3_d, g_4_d, g_5_d, g_6_d,
                                    g_7_d, g_8_d, g_9_d, g_10_d, g_11_d, g_12_d,
                                    g_13_d, g_14_d, g_15_d, g_16_d, g_17_d, 
                                    g_18_d );

    if( proc == master ) fprintf( stdout, "Data initialization completed.\n" );
    hipDeviceSynchronize();
    hipEventRecord( stopSetup, 0 );
    hipEventSynchronize( stopSetup );

//====================== INTERFACE RELAXATION LOOP =============================
    RELAX_GRAV = 0.f;
    hipMemcpyToSymbol(HIP_SYMBOL( grav_d), &RELAX_GRAV, sizeof(float), 0, hipMemcpyHostToDevice );
    for( step = 1; step <= relaxStep; step++){

        update_phi <<<dimgrid,dimblock>>> ( phi_d, f_0_d, f_1_d, f_2_d, f_3_d,
                                            f_4_d, f_5_d, f_6_d );

        update_rho <<<dimgrid,dimblock>>> ( rho_d, g_0_d, g_1_d, g_2_d, g_3_d, 
                                            g_4_d, g_5_d, g_6_d, g_7_d, g_8_d, 
                                            g_9_d, g_10_d, g_11_d, g_12_d, 
                                            g_13_d, g_14_d, g_15_d, g_16_d, 
                                            g_17_d, g_18_d );

        update_velocity <<<dimgrid,dimblock>>> ( rho_d, ux_d, uy_d, uz_d, g_1_d, 
                                                 g_2_d, g_3_d, g_4_d, g_5_d, 
                                                 g_6_d, g_7_d, g_8_d, g_9_d, 
                                                 g_10_d, g_11_d, g_12_d, g_13_d, 
                                                 g_14_d, g_15_d, g_16_d, g_17_d,
                                                 g_18_d );

        // Update phi values in the ghost nodes for the Laplacian calculation 
        pack_mpi_phi <<<dimgrid,dimblock>>> ( topF_snd_d, botF_snd_d, phi_d );
   	
        hipDeviceSynchronize();
        hipMemcpy( topF_snd, topF_snd_d, sizeof(float)*bufSize, hipMemcpyDeviceToHost );
        hipMemcpy( botF_snd, botF_snd_d, sizeof(float)*bufSize, hipMemcpyDeviceToHost );
        mpiUpdate_phi( topF_snd, botF_snd, topF_rcv, botF_rcv );

        hipMemcpy( topF_rcv_d, topF_rcv, sizeof(float)*bufSize, hipMemcpyHostToDevice );
        hipMemcpy( botF_rcv_d, botF_rcv, sizeof(float)*bufSize, hipMemcpyHostToDevice );
        unpack_mpi_phi <<<dimgrid,dimblock>>> ( topF_rcv_d, botF_rcv_d, phi_d );

        collision_f <<<dimgrid,dimblock>>> ( nb_east_d, nb_west_d, nb_north_d, 
                                             nb_south_d, phi_d, rho_d, ux_d, 
                                             uy_d, uz_d, f_0_d, f_1_d, f_2_d,
                                             f_3_d, f_4_d, f_5_d, f_6_d );

        collision_g <<<dimgrid,dimblock>>> ( nb_east_d, nb_west_d, nb_north_d, 
                                             nb_south_d, phi_d, rho_d, ux_d, 
                                             uy_d, uz_d,  g_0_d, g_1_d, g_2_d, 
                                             g_3_d, g_4_d, g_5_d, g_6_d,
                                             g_7_d, g_8_d, g_9_d, g_10_d,
                                             g_11_d, g_12_d, g_13_d, 
                                             g_14_d, g_15_d, g_16_d, 
                                             g_17_d, g_18_d );

        // Before streaming f we need to make sure that the outward f components
        // in the ghost nodes are correct, because they are required in the 
        // smoothing step introduced by ZSC.
        // PERFORMANCE NOTE: see effect of skipping smoothing step for these 
        //                   nodes in order to save one MPI exchange.
        pack_mpi_f <<<dimgrid,dimblock>>> ( topF_snd_d, botF_snd_d, f_5_d, f_6_d );
   	
        hipDeviceSynchronize();
        hipMemcpy( topF_snd, topF_snd_d, sizeof(float)*bufSize, hipMemcpyDeviceToHost );
        hipMemcpy( botF_snd, botF_snd_d, sizeof(float)*bufSize, hipMemcpyDeviceToHost );
        mpiUpdate_f( topF_snd, botF_snd, topF_rcv, botF_rcv );

        hipMemcpy( topF_rcv_d, topF_rcv, sizeof(float)*bufSize, hipMemcpyHostToDevice );
        hipMemcpy( botF_rcv_d, botF_rcv, sizeof(float)*bufSize, hipMemcpyHostToDevice );
        unpack_mpi_f <<<dimgrid,dimblock>>> ( topF_rcv_d, botF_rcv_d, f_5_d, f_6_d );

        stream_f <<<dimgrid,dimblock>>> ( nb_east_d, nb_west_d, nb_north_d,
                                          nb_south_d, f_0_d, f_1_d, f_2_d,
                                          f_3_d, f_4_d, f_5_d, f_6_d );

        stream_g <<<dimgrid,dimblock>>> ( nb_east_d, nb_west_d, nb_north_d,
                                          nb_south_d, g_0_d,  g_1_d, g_2_d,
                                          g_3_d, g_4_d, g_5_d, g_6_d, g_7_d,
                                          g_8_d, g_9_d, g_10_d, g_11_d, g_12_d,
                                          g_13_d, g_14_d, g_15_d, g_16_d, 
                                          g_17_d, g_18_d );

        // Think carefully how to do this block partition to avoid doing
        // multiple if branches all over the place
        // This is the part that really hurts performance: copying to the host 
        // and back to do the MPI exchange. We should use streams and asynchronous
        // exchange of the boundaries only so that we can work on the core nodes
        // while the host/device and host/host communication is going on. 
        pack_mpi <<<dimgrid,dimblock>>> ( top_snd_d, bot_snd_d,
                                          f_5_d,  f_6_d,  g_5_d,  g_6_d,  
                                          g_11_d, g_12_d, g_13_d, g_14_d, 
                                          g_15_d, g_16_d, g_17_d, g_18_d );

        hipDeviceSynchronize();
        hipMemcpy( top_snd, top_snd_d, sizeof(float)*bufSize*6, hipMemcpyDeviceToHost );
        hipMemcpy( bot_snd, bot_snd_d, sizeof(float)*bufSize*6, hipMemcpyDeviceToHost );
        mpiUpdate( top_snd, bot_snd, top_rcv, bot_rcv );

        hipMemcpy( top_rcv_d, top_rcv, sizeof(float)*bufSize*6, hipMemcpyHostToDevice );
        hipMemcpy( bot_rcv_d, bot_rcv, sizeof(float)*bufSize*6, hipMemcpyHostToDevice );
        unpack_mpi <<<dimgrid,dimblock>>> ( top_rcv_d, bot_rcv_d,
                                            f_5_d,  f_6_d,  g_5_d,  g_6_d,  
                                            g_11_d, g_12_d, g_13_d, g_14_d, 
                                            g_15_d, g_16_d, g_17_d, g_18_d );

        if( (step%stat) == 0 ){
            hipDeviceSynchronize();
            hipMemcpy( phi, phi_d, sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( ux,  ux_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( uy,  uy_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( uz,  uz_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            stats( phi, ux, uy, uz );
        }
        if( (step%save) == 0 ){
            hipDeviceSynchronize();
            hipMemcpy( phi, phi_d, sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( rho, rho_d, sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( ux,  ux_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( uy,  uy_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( uz,  uz_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            vtkSave( nb_east, nb_west, nb_north, nb_south, phi, rho, ux, uy, uz );
        }
    }
    logSave( devName, devMem, nb_east, nb_west, nb_north, nb_south, 
             phi, rho, ux, uy, uz );
    if( proc == master ) fprintf( stdout, "Relaxation run completed.\n" );
    RELAX = 0;
    hipMemcpyToSymbol(HIP_SYMBOL( grav_d), &grav, sizeof(float), 0, hipMemcpyHostToDevice );

//====================== SAVE RELAXED CONFIGURATION ============================
    hipDeviceSynchronize();
    hipMemcpy( phi, phi_d, sizeof(float)*gridSize, hipMemcpyDeviceToHost );
    hipMemcpy( rho, rho_d, sizeof(float)*gridSize, hipMemcpyDeviceToHost );
    hipMemcpy( ux,  ux_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
    hipMemcpy( uy,  uy_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
    hipMemcpy( uz,  uz_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
    step = 0;
    stats( phi, ux, uy, uz );
    vtkSave( nb_east, nb_west, nb_north, nb_south, phi, rho, ux, uy, uz );
    hipEventRecord( stopRelax, 0 );
    hipEventSynchronize( stopRelax );

//====================== MAIN CALCULATION LOOP =================================
    if( proc == master ) printf( "Starting evolution run ...\n" );
    for( step = 1; step < maxStep; step++){

        update_phi <<<dimgrid,dimblock>>> ( phi_d, f_0_d, f_1_d, f_2_d, f_3_d,
                                            f_4_d, f_5_d, f_6_d );

        update_rho <<<dimgrid,dimblock>>> ( rho_d, g_0_d, g_1_d, g_2_d, g_3_d, 
                                            g_4_d, g_5_d, g_6_d, g_7_d, g_8_d, 
                                            g_9_d, g_10_d, g_11_d, g_12_d, 
                                            g_13_d, g_14_d, g_15_d, g_16_d, 
                                            g_17_d, g_18_d );

        update_velocity <<<dimgrid,dimblock>>> ( rho_d, ux_d, uy_d, uz_d, g_1_d, 
                                                 g_2_d, g_3_d, g_4_d, g_5_d, 
                                                 g_6_d, g_7_d, g_8_d, g_9_d, 
                                                 g_10_d, g_11_d, g_12_d, g_13_d, 
                                                 g_14_d, g_15_d, g_16_d, g_17_d,
                                                 g_18_d );

        // Update phi values in the ghost nodes for the Laplacian calculation 
        pack_mpi_phi <<<dimgrid,dimblock>>> ( topF_snd_d, botF_snd_d, phi_d );

        hipDeviceSynchronize();
        hipMemcpy( topF_snd, topF_snd_d, sizeof(float)*bufSize, hipMemcpyDeviceToHost );
        hipMemcpy( botF_snd, botF_snd_d, sizeof(float)*bufSize, hipMemcpyDeviceToHost );
        mpiUpdate_phi( topF_snd, botF_snd, topF_rcv, botF_rcv );

        hipMemcpy( topF_rcv_d, topF_rcv, sizeof(float)*bufSize, hipMemcpyHostToDevice );
        hipMemcpy( botF_rcv_d, botF_rcv, sizeof(float)*bufSize, hipMemcpyHostToDevice );
        unpack_mpi_phi <<<dimgrid,dimblock>>> ( topF_rcv_d, botF_rcv_d, phi_d );

        collision_f <<<dimgrid,dimblock>>> ( nb_east_d, nb_west_d, nb_north_d,
                                             nb_south_d, phi_d, rho_d, ux_d, 
                                             uy_d, uz_d, f_0_d, f_1_d, f_2_d,
                                             f_3_d, f_4_d, f_5_d, f_6_d );

        collision_g <<<dimgrid,dimblock>>> ( nb_east_d, nb_west_d, nb_north_d, 
                                             nb_south_d, phi_d, rho_d, ux_d, 
                                             uy_d, uz_d,  g_0_d, g_1_d, g_2_d, 
                                             g_3_d, g_4_d, g_5_d, g_6_d,
                                             g_7_d, g_8_d, g_9_d, g_10_d,
                                             g_11_d, g_12_d, g_13_d, 
                                             g_14_d, g_15_d, g_16_d, 
                                             g_17_d, g_18_d );

        // Before streaming f we need to make sure that the outward f components
        // in the ghost nodes are correct, because they are required in the 
        // smoothing step introduced by ZSC.
        // PERFORMANCE NOTE: see effect of skipping smoothing step for these 
        //                   nodes in order to save one MPI exchange.
        pack_mpi_f <<<dimgrid,dimblock>>> ( topF_snd_d, botF_snd_d, f_5_d, f_6_d );

        hipDeviceSynchronize();
        hipMemcpy( topF_snd, topF_snd_d, sizeof(float)*bufSize, hipMemcpyDeviceToHost );
        hipMemcpy( botF_snd, botF_snd_d, sizeof(float)*bufSize, hipMemcpyDeviceToHost );
        mpiUpdate_f( topF_snd, botF_snd, topF_rcv, botF_rcv );

        hipMemcpy( topF_rcv_d, topF_rcv, sizeof(float)*bufSize, hipMemcpyHostToDevice );
        hipMemcpy( botF_rcv_d, botF_rcv, sizeof(float)*bufSize, hipMemcpyHostToDevice );
        unpack_mpi_f <<<dimgrid,dimblock>>> ( topF_rcv_d, botF_rcv_d, f_5_d, f_6_d );


        stream_f <<<dimgrid,dimblock>>> ( nb_east_d, nb_west_d, nb_north_d, 
                                          nb_south_d, f_0_d, f_1_d, f_2_d, 
                                          f_3_d, f_4_d, f_5_d, f_6_d );

        stream_g <<<dimgrid,dimblock>>> ( nb_east_d, nb_west_d, nb_north_d,
                                          nb_south_d, g_0_d, g_1_d, g_2_d,  
                                          g_3_d, g_4_d, g_5_d, g_6_d, g_7_d, 
                                          g_8_d, g_9_d, g_10_d, g_11_d, g_12_d,
                                          g_13_d, g_14_d, g_15_d, g_16_d, 
                                          g_17_d, g_18_d );

        // Think carefully how to do this block partition to avoid doing
        // multiple if branches all over the place
        // This is the part that really hurts performance: copying to the host 
        // and back to do the MPI exchange. We should use streams and asynchronous
        // exchange of the boundaries only so that we can work on the core nodes
        // while the host/device and host/host communication is going on. 
        pack_mpi <<<dimgrid,dimblock>>> ( top_snd_d, bot_snd_d,
                                          f_5_d,  f_6_d,  g_5_d,  g_6_d,  
                                          g_11_d, g_12_d, g_13_d, g_14_d, 
                                          g_15_d, g_16_d, g_17_d, g_18_d );

        hipDeviceSynchronize();
        hipMemcpy( top_snd, top_snd_d, sizeof(float)*bufSize*6, hipMemcpyDeviceToHost );
        hipMemcpy( bot_snd, bot_snd_d, sizeof(float)*bufSize*6, hipMemcpyDeviceToHost );
        mpiUpdate( top_snd, bot_snd, top_rcv, bot_rcv );

        hipMemcpy( top_rcv_d, top_rcv, sizeof(float)*bufSize*6, hipMemcpyHostToDevice );
        hipMemcpy( bot_rcv_d, bot_rcv, sizeof(float)*bufSize*6, hipMemcpyHostToDevice );
        unpack_mpi <<<dimgrid,dimblock>>> ( top_rcv_d, bot_rcv_d,
                                            f_5_d,  f_6_d,  g_5_d,  g_6_d,  
                                            g_11_d, g_12_d, g_13_d, g_14_d, 
                                            g_15_d, g_16_d, g_17_d, g_18_d );


        if( (step%stat) == 0 ){
            hipDeviceSynchronize();
            hipMemcpy( phi, phi_d, sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( ux,  ux_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( uy,  uy_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( uz,  uz_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            stats( phi, ux, uy, uz );
        }
        if( (step%save) == 0 ){
            hipDeviceSynchronize();
            hipMemcpy( phi, phi_d, sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( rho, rho_d, sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( ux,  ux_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( uy,  uy_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            hipMemcpy( uz,  uz_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
            vtkSave( nb_east, nb_west, nb_north, nb_south, phi, rho, ux, uy, uz );
        }

    }
    if( proc == master ) fprintf( stdout, "Evolution run completed.\n" );
    hipDeviceSynchronize();
    hipEventRecord( stopMain, 0 );
    hipEventSynchronize( stopMain );	
    hipEventElapsedTime( &setupTime,   start,     stopSetup );
    hipEventElapsedTime( &relaxTime,   stopSetup, stopRelax );
    hipEventElapsedTime( &mainTime,    stopRelax, stopMain );
    hipEventElapsedTime( &elapsedTime, start,     stopMain );
    if( proc == master ){
        printf( "Setup time      : %6.3f ms\n", setupTime );
        printf( "Relaxation time : %6.3f ms\n", relaxTime );
        printf( "Evolution time  : %6.3f ms\n", mainTime );
        printf( "Total time      : %6.3f ms\n", elapsedTime );
        printf( "Time for lb step: %6.3f ms\n", mainTime/( 1.f*maxStep ) );
    }

    // Save final configuration
    hipMemcpy( phi, phi_d, sizeof(float)*gridSize, hipMemcpyDeviceToHost );
    hipMemcpy( rho, rho_d, sizeof(float)*gridSize, hipMemcpyDeviceToHost );
    hipMemcpy( ux,  ux_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
    hipMemcpy( uy,  uy_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
    hipMemcpy( uz,  uz_d,  sizeof(float)*gridSize, hipMemcpyDeviceToHost );
    logSave( devName, devMem, nb_east, nb_west, nb_north, nb_south, 
             phi, rho, ux, uy, uz );
    stats( phi, ux, uy, uz );
    vtkSave( nb_east, nb_west, nb_north, nb_south, phi, rho, ux, uy, uz );

//====================== Free host dynamic arrays ==============================
    free( phi );       free( rho );
    free( nb_north );  free( nb_south );
    free( nb_east );   free( nb_west );   free( ux );  free( uy );  free( uz );

//====================== Free device dynamic arrays ==============================
    hipFree( nb_north_d ); hipFree( nb_south_d );
    hipFree( nb_east_d );   hipFree( nb_west_d );

    hipFree( phi_d ); hipFree( rho_d ); 
    hipFree( ux_d );  hipFree( uy_d );  hipFree( uz_d );

    hipFree( f_0_d ); hipFree( f_1_d ); hipFree( f_2_d ); hipFree( f_3_d );
    hipFree( f_4_d ); hipFree( f_5_d ); hipFree( f_6_d );

    hipFree( g_0_d );  hipFree( g_1_d );  hipFree( g_2_d );
    hipFree( g_3_d );  hipFree( g_4_d );  hipFree( g_5_d );
    hipFree( g_6_d );  hipFree( g_7_d );  hipFree( g_8_d );
    hipFree( g_9_d );  hipFree( g_10_d ); hipFree( g_11_d );
    hipFree( g_12_d ); hipFree( g_13_d ); hipFree( g_14_d );
    hipFree( g_15_d ); hipFree( g_16_d ); hipFree( g_17_d ); 
    hipFree( g_18_d );

    MPI_Finalize();

    return 0;
}



